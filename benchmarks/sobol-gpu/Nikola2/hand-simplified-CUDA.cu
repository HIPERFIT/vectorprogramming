#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <inttypes.h>
extern "C" __global__ void kernel0(int32_t* input, int32_t input_length,
                                   double* output,
                                   int32_t output_length, int32_t output_size)
{
    for (int32_t j = blockIdx.x * blockDim.x + threadIdx.x;
         j < output_size;
         j += blockDim.x * gridDim.x) {
        uint32_t xa = 0U;
        uint8_t lastTestBit = 1;
  
        for (int i = 0; i < 30; ++i) {
            uint32_t n;
            uint8_t testBit;

            // j % (x_04 > (input_length > x_04 ? x_04 : input_length) ? input_length > x_04 ? x_04 : input_length : x_04)
            // (u > (t > u ? u : t) ? (t > u ? u : t) : u)
            // <=>
            // (u > min(u,t) ? min(u,t) : u)
            // <=>
            // min(u,t)
            n = (uint32_t) input[j % min(input_length, output_size)]; // this should be hoisted out of the loop (does nvcc do it?)

            uint32_t grayCode = n ^ (n >> 1U);
            testBit = (grayCode & 1 << i) != 0;

            if (testBit) {
                uint32_t v;
                v = 1U << 29U - (uint32_t) i; // direction numbers (these should have been arguments to the kernel)
                if (lastTestBit) {
                    xa = v ^ xa;
                } else {
                    xa = v;
                }
                lastTestBit = testBit;
            }
        }

        output[j] = (double) (int32_t) xa / (double) (1 << 30);
    }
}
